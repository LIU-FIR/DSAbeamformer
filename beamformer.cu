#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <iomanip>
#include <string>
#include <hipblas.h>
#include <cmath>
// #include <thrust::constant_iterator.h>
#include <fstream>
#include "bitmap_image.hpp" // colorscheme

// DSA CONSTANTS
#define N_BEAMS 256
#define N_ANTENNAS 64
#define N_FREQUENCIES 256
#define N_AVERAGING 16
#define N_POL 2
#define N_TIMESTEPS_PER_CALL 1*N_AVERAGING*N_POL

#define N_CX 2
#define N_BLOCKS_on_GPU 4
#define BYTES_PER_GEMM  N_ANTENNAS*N_FREQUENCIES*N_TIMESTEPS_PER_CALL

// Data Indexing, Offsets
#define N_GPUS 8
#define TOT_CHANNELS 2048
#define START_F 1.28
#define END_F 1.53
#define ZERO_PT 0

// Numerical Constants
#define C_SPEED 299792458.0
#define PI 3.14159265358979


// Type Constants
#define N_BITS 8
#define MAX_VAL 127

#define SIG_BITS 4
#define SIG_MAX_VAL 7


// nvcc beamformer.cu -o beam -lcublas -lsfml-graphics


#define DEG2RAD(x) ((x)*PI/180.0)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}


hipEvent_t start;
hipEvent_t stop;

#define START_TIMER() {                         \
	gpuErrchk(hipEventCreate(&start));       \
	gpuErrchk(hipEventCreate(&stop));        \
	gpuErrchk(hipEventRecord(start));        \
}

#define STOP_RECORD_TIMER(name) {                           \
	gpuErrchk(hipEventRecord(stop));                     \
	gpuErrchk(hipEventSynchronize(stop));                \
	gpuErrchk(hipEventElapsedTime(&name, start, stop));  \
	gpuErrchk(hipEventDestroy(start));                   \
	gpuErrchk(hipEventDestroy(stop));                    \
}



typedef char2 CxInt8_t;
typedef char char4_t[4]; //32-bit so global memory bandwidth usage is optimal
typedef char char8_t[8]; //64-bit so global memory bandwidth usage is optimal
typedef CxInt8_t cuChar4_t[4];



__global__
void detect_sum(hipComplex *input, float *output){
	/*
	Sum over N_TIMESTEPS_PER_CALL
	number of threads = N_BEAMS = blockDim.x
	number of blocks = N_FREQUENCIES
	*/
	__shared__ float shmem[N_BEAMS];

	int input_idx  = blockIdx.x * N_BEAMS * N_TIMESTEPS_PER_CALL + threadIdx.x;
	int local_idx  = threadIdx.x; // which beam
	int output_idx = blockIdx.x * N_BEAMS + threadIdx.x;

	hipComplex in;

	// #pragma unroll
	for (int i = 0; i < N_TIMESTEPS_PER_CALL; i++){
		in = input[input_idx];
		shmem[local_idx] += in.x*in.x;// + in.y*in.y;
		input_idx += N_BEAMS; // go to the next time step
	}

	output[output_idx] = shmem[local_idx]; // slowest to fastest indicies: freq, beam
}



__global__
void expand_input(char *input, char *output, int input_size){
	/*
	This code takes in an array of 4-bit integers and returns an array of 8-bit integers.
	To maximize global memory bandwidth and symplicity, two special char types are 
	defined: char4_t and char8_t. The size of these types are 32-bit and 64-bit respectively. These
	enable coalesced memory accesses, but then require the gpu to handle the 4 to 8-bit
	conversion simultaneously for 8 numbers (4 real/imaginary pairs). 
	*/

	__shared__ float shmem_in[32];
	__shared__ double shmem_out[32];

	char4_t *char_shmem_in;
	cuChar4_t *char_shmem_out;

	char_shmem_in = reinterpret_cast<char4_t *>(shmem_in);
	char_shmem_out = reinterpret_cast<cuChar4_t *>(shmem_out);

	int local_idx = threadIdx.x;
	int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

	while (global_idx < input_size/sizeof(float)){
		shmem_in[local_idx] = ((float *) input)[global_idx]; // read eight pieces of 4-bit memory into shared memory

		//#pragma unroll
		for (int i = 0; i < 4; i++){
			char temp = char_shmem_in[local_idx][i];

			// break the char into two 4-bit chunks, then convert to 8-bit
			char high = (temp >> 4); // roll the 		most  significant 4 bits over the least significant 4 bits
			char low = (temp << 4);  // roll the 		least significant 4 bits over the most  significant 4 bits
			low = (low >> 4); 	 	 // roll the *new* 	most  significant 4 bits over the least significant 4 bits

			// store the two 8-bit numbers to the output shared memory array
			char_shmem_out[local_idx][i].x = high; 
			char_shmem_out[local_idx][i].y = low;
		}

		((double *) output)[global_idx] = shmem_out[local_idx];	// write eight pieces of 8-bit memory out to global memory

		global_idx += gridDim.x * blockDim.x;
	}
}




int main(){
	std::cout << "hello" << std::endl;

	std::ofstream f;
	f.open("data.py");
	f << "A = [[";

	int N_DIRS = 1024;

	int A_rows	 = N_BEAMS;
	int A_cols 	 = N_ANTENNAS;
	int A_stride = A_rows*A_cols;
	int B_cols	 = N_TIMESTEPS_PER_CALL;
	int B_rows	 = A_cols;
	int B_stride = B_rows*B_cols;
	int C_rows	 = A_rows;
	int C_cols	 = B_cols;
	int C_stride = C_rows*C_cols;
	float bw_per_channel = (END_F - START_F)/TOT_CHANNELS; 

	/* GPU Variables */
	CxInt8_t *d_A; 				// Weight matrix (N_BEAMS X N_ANTENNAS, for N_FREQUENCIES)
	CxInt8_t *d_B; 				// Data Matrix (N_ANTENNAS X N_TIMESTEPS_PER_CALL, for N_FREQUENCIES)
	char *d_data;			// Raw input data (Before data massaging)
	hipComplex *d_C;				// Beamformed output (N_BEAMS X N_TIMESTEPS_PER_CALL, for N_FREQUENCIES)
	float *d_out;			// Data after being averaged over 16 time samples and 2 polarizations
	float *d_dedispersed;	// Data after being de-dispersed
	float *d_vec_ones;		// Vector of all ones for de-dispersion

	/* HOST Variables */
	CxInt8_t *A = new CxInt8_t[A_cols*A_rows*N_FREQUENCIES];
	CxInt8_t *B = new CxInt8_t[B_cols*B_rows*N_FREQUENCIES];
	char *data = new char[BYTES_PER_GEMM*N_BLOCKS_on_GPU]; //should be the size of one "dada block", data is 4-bit so real/imag is packed into one 8-bit char
	float *out_dedispersed = new float[N_BEAMS];
	float *vec_ones = new float[N_FREQUENCIES];

	// thrust::constant_iterator<float> vec_o(1)

	float* pos = new float[N_ANTENNAS];		// Locations of antennas
	float* dir = new float[N_BEAMS];		// Direction of beams
	int gpu = 0;							// Unique identifier for each GPU


	/* Populate location/direction Matricies */
	for (int i = 0; i < N_ANTENNAS; i++){
		pos[i] = i*500.0/(N_ANTENNAS-1) - 250.0;
	}

	/* Directions for Beamforming */
	for (int i = 0; i < N_BEAMS; i++){
		dir[i] = i*DEG2RAD(7.0)/(N_BEAMS-1) - DEG2RAD(3.5);
	}

	/* Create vector of ones for Dedispersion */
	for (int i = 0; i < N_FREQUENCIES; i++){
		vec_ones[i] = 1.0;
	}


	// Fourier Coefficient Matrix
	for (int i = 0; i < N_FREQUENCIES; i++){
		float freq = END_F - (ZERO_PT + gpu*TOT_CHANNELS/(N_GPUS-1) + i)*bw_per_channel;
		float wavelength = C_SPEED/(1E9*freq);
		for (int j = 0; j < N_ANTENNAS; j++){
			for (int k = 0; k < N_BEAMS; k++){
				A[i*A_stride + j*N_BEAMS + k].x = round(MAX_VAL*cos(-2*PI*pos[j]*sin(dir[k])/wavelength));
				A[i*A_stride + j*N_BEAMS + k].y = round(MAX_VAL*sin(-2*PI*pos[j]*sin(dir[k])/wavelength));
			}
		}
		// std::cout << "A[] = " << (int) A[i*N_ANTENNAS*N_BEAMS].x << "+"<< (int) A[i*N_ANTENNAS*N_BEAMS].y << "j" << std::endl;
	}

	// Signal Matrix
	// int test_frequency = 10;
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	/* Allocate and Move Memory to Device */
	hipMalloc(&d_A, 	A_rows*A_cols*N_FREQUENCIES*sizeof(CxInt8_t));
	hipMalloc(&d_B, 	B_rows*B_cols*N_FREQUENCIES*sizeof(CxInt8_t));
	hipMalloc(&d_C, 	C_rows*C_cols*N_FREQUENCIES*sizeof(hipComplex));
	hipMalloc(&d_data, BYTES_PER_GEMM*N_BLOCKS_on_GPU);
	hipMalloc(&d_out,  N_BEAMS*N_FREQUENCIES * sizeof(float));
	hipMalloc(&d_dedispersed, N_BEAMS*sizeof(float));
	hipMalloc(&d_vec_ones, N_BEAMS*sizeof(float));


	hipMemcpy(d_A, A, A_rows*A_cols*N_FREQUENCIES*sizeof(CxInt8_t), hipMemcpyHostToDevice);
	hipMemcpy(d_vec_ones, vec_ones, N_FREQUENCIES*sizeof(float), hipMemcpyHostToDevice);


	for (int iii = 0; iii < N_DIRS; iii++){
		float test_direction = DEG2RAD(-3.5) + iii*DEG2RAD(7.0)/(N_DIRS-1);

		// for (int i = 0; i < N_FREQUENCIES; i++){
		// 	float freq = END_F - (ZERO_PT + gpu*TOT_CHANNELS/(N_GPUS-1) + i)*bw_per_channel;
		// 	float wavelength = C_SPEED/(1E9*freq);

		// 	for (int j = 0; j < N_TIMESTEPS_PER_CALL; j++){
		// 		for (int k = 0; k < N_ANTENNAS; k++){
		// 			if (i == test_frequency){
		// 				B[i*N_TIMESTEPS_PER_CALL*N_ANTENNAS + j*N_ANTENNAS + k].x = round(MAX_VAL*cos(2*PI*pos[k]*sin(test_direction)/wavelength));
		// 				B[i*N_TIMESTEPS_PER_CALL*N_ANTENNAS + j*N_ANTENNAS + k].y = round(MAX_VAL*sin(2*PI*pos[k]*sin(test_direction)/wavelength));
		// 			}
		// 		}
		// 	}

		// }
	 	
		// int simulated_direction = 100;
		int current_block = 0;
		// int tot_avging = N_POL*N_AVERAGING;

		char high, low;

		for (int i = 0; i < N_FREQUENCIES; i++){
			float freq = END_F - (ZERO_PT + gpu*TOT_CHANNELS/(N_GPUS-1) + i)*bw_per_channel;
			// std::cout << "freq: " << freq << std::endl;
			float wavelength = C_SPEED/(1E9*freq);
			for (int j = 0; j < N_TIMESTEPS_PER_CALL; j++){
				for (int k = 0; k < N_ANTENNAS; k++){

					high = ((char) round(SIG_MAX_VAL*cos(2*PI*pos[k]*sin(test_direction)/wavelength))); //real
					low  = ((char) round(SIG_MAX_VAL*sin(2*PI*pos[k]*sin(test_direction)/wavelength))); //imag

					data[i*B_stride + j*N_ANTENNAS + k] = (high << 4) | (0x0F & low);
				}
			}
		}



		// hipMemcpy(d_B, B, B_rows*B_cols*N_FREQUENCIES*sizeof(CxInt8_t), hipMemcpyHostToDevice);
		hipMemcpy(&(d_data[BYTES_PER_GEMM*current_block]), data, BYTES_PER_GEMM, hipMemcpyHostToDevice);

		
		expand_input<<<1000, 32>>>(d_data, (char *) d_B, B_stride*N_FREQUENCIES);



		// Multiplicative Constants
		hipComplex inv_max_value, zero;//, one;
		inv_max_value.x = 1.0/MAX_VAL;
		inv_max_value.y = 0;
		zero.x = 0;
		zero.y = 0;
		// one.x = 1;
		// one.y = 0;

		hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
									A_rows, B_cols, A_cols,
									&inv_max_value,
									d_A, HIP_C_8I, A_rows, A_stride,
									d_B, HIP_C_8I, B_rows, B_stride,
									&zero,
									d_C, HIP_C_32F, C_rows, C_stride,
									N_FREQUENCIES, HIP_C_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);


		detect_sum<<<N_FREQUENCIES, N_BEAMS>>>(d_C, d_out);

		float f_one = 1.0;
		float f_zero = 0.0;


		hipblasSgemv(handle, HIPBLAS_OP_N,
					N_BEAMS, N_FREQUENCIES,
					&f_one,
					d_out, N_BEAMS,
					d_vec_ones, 1,
					&f_zero,
					d_dedispersed, 1);


		//gemv to dedisperse
		//copy to host
		//sfml image

		hipMemcpy(out_dedispersed, d_dedispersed, N_BEAMS*sizeof(float), hipMemcpyDeviceToHost);

		for (int i = 0; i < N_BEAMS; i++){
			f << out_dedispersed[i];
			if (i != N_BEAMS - 1){
				f << ",";
			}
		}

		if (iii != N_DIRS-1){
			f << "],\n[";
		} else {
			f<< "]]";
		}

	}

	#if 0
		float max = 0;
		float rms = 0.0;
		int max_i = 0;
		for (int i = 0; i < 256; i++){
			if (C[i].x>max){
				rms += C[i].x*C[i].x;
				max = C[i].x;
				max_i = i;
			}
			std::cout << "C[" << i <<"] = " << C[i].x << "+" << C[i].y << "j" << std::endl;
		}

		std::cout << "max(C) = " << max_i << ", " << max << std::endl;
		std::cout << "rms(c) = " << sqrt(rms/256.0) << std::endl;
	#endif


	f.close();


	hipFree(d_A);
	hipFree(d_C);
	hipFree(d_B);
	hipFree(d_data);
	hipFree(d_out);
	hipFree(d_dedispersed);
	hipFree(d_vec_ones);

	delete[] vec_ones;
	delete[] A;
	delete[] out_dedispersed;
	delete[] data;
	delete[] B;
	delete[] pos;
	delete[] dir;

	hipblasDestroy(handle);
	return 0;
}