#include <iostream>
#include <hipblas.h>


int main(){
	
	std::cout << "hello" << std::endl;

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipComplex temp;
	temp.x = 0;
	temp.y = 3;
	std::cout << "test:" << temp.x << std::endl;


	HIP_R_16F te = 12.4;
	std::cout << "rt " << te << std::endl;

	hipblasDestroy(handle);
	return 0;
}