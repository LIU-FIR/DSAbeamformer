#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>


#include <thrust/transform.h>
#include <thrust/for_each.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h>

hipEvent_t start;
hipEvent_t stop;

#define START_TIMER() {                         \
	gpuErrchk(hipEventCreate(&start));       \
	gpuErrchk(hipEventCreate(&stop));        \
	gpuErrchk(hipEventRecord(start));        \
}

#define STOP_RECORD_TIMER(name) {                           \
	gpuErrchk(hipEventRecord(stop));                     \
	gpuErrchk(hipEventSynchronize(stop));                \
	gpuErrchk(hipEventElapsedTime(&name, start, stop));  \
	gpuErrchk(hipEventDestroy(start));                   \
	gpuErrchk(hipEventDestroy(stop));                    \
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

void gpuBLASchk(int errval){
	if (errval != HIPBLAS_STATUS_SUCCESS){
		std::cerr << "Failed BLAS call, error code " << errval << std::endl;
	}
}


struct detect_functor{
	__host__ __device__
	hipComplex operator() (hipComplex &b){
		hipComplex a;
		a.y = b.y;
		a.x = b.x*b.x;
		return a;
	}
};

struct B_detect_functor{
	__host__ __device__
	void operator() (hipComplex &b){
		b.x *= b.x;
	}
};


struct C_detect_functor{
	__host__ __device__
	float operator() (hipComplex &b){
		return b.x*b.x;
	}
};


int main(){
	
	std::cout << "hello " << std::endl;

	hipblasHandle_t handle;
	gpuBLASchk(hipblasCreate(&handle));

	
	int n_timesteps = 16;
	int n_pol = 2;
	int n_outputs_per_input = 2;
	int n_time_pol_out = n_timesteps*n_pol*n_outputs_per_input;
	int n_beams = 256;
	int n_freq = 256;


	std::cout << "n_timesteps = " << n_timesteps << std::endl;
	std::cout << "n_pol = " << n_pol << std::endl;
	std::cout << "n_outputs_per_input = " << n_outputs_per_input << std::endl;
	std::cout << "n_time_pol_out = " << n_time_pol_out << std::endl;
	std::cout << "n_beams = " << n_beams << std::endl;
	std::cout << "n_freq = " << n_freq << std::endl;

	int N = n_time_pol_out*n_beams*n_freq;
	std::cout  << "N = " << N << std::endl;


	hipStream_t s1, s2;
	gpuErrchk(hipStreamCreate(&s1));
	gpuErrchk(hipStreamCreate(&s2));
	// int N = 10000000;

	hipComplex *data = new hipComplex[N];

	for (int i = 0; i < N; i++){
		data[i].y = 0;
		data[i].x = i + .7;
	}

	for (int i = 0; i < 10; i ++){
		std::cout << "x[" << i << "] = " << data[i].x << " + " << data[i].y << "j" << std::endl;
	}
	std::cout << std::endl;

	hipComplex *d_data, *d_data2, *detect;

	gpuErrchk(hipMalloc(&d_data, N*sizeof(hipComplex)));
	gpuErrchk(hipMalloc(&d_data2, N*sizeof(hipComplex)));
	gpuErrchk(hipMalloc(&detect, n_beams*n_freq*n_outputs_per_input*sizeof(hipComplex)));

	gpuErrchk(hipMemcpy(d_data, data, N*sizeof(hipComplex), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_data2, data, N*sizeof(hipComplex), hipMemcpyHostToDevice));

	// gpuErrchk(hipMalloc(&detect, n_beams*n_freq*n_outputs_per_input*sizeof(hipComplex)));


	// thrust::device_ptr<hipComplex> dev_data(d_data);
	// thrust::device_ptr<hipComplex> dev_data2(d_data2);


	// float ms_timer = 0;
	// START_TIMER();
	// thrust::for_each(thrust::cuda::par.on(s1), dev_data, dev_data+N, B_detect_functor());
	// thrust::for_each(thrust::cuda::par.on(s2), dev_data2, dev_data2+N, B_detect_functor());
	// STOP_RECORD_TIMER(ms_timer);

	// gpuErrchk(hipMemcpy(data, d_data, N*sizeof(hipComplex), hipMemcpyDeviceToHost));

	// for (int i = 0; i < 10; i ++){
	// 	std::cout << "x[" << i << "] = " << data[i].x << " + " << data[i].y << "j" << std::endl;
	// }

	// std::cout << N <<" data points in " << ms_timer	 << " ms" << std::endl;

	int f = 0;
	int o = 0;

	// hipDeviceSynchronize();

	gpuBLASchk(hipblasCdotc(handle, 10,
							d_data, 1,
							d_data2, 1,
							detect));

	// gpuBLASchk(hipblasCdotc(handle, n_timesteps*n_pol,
	// 						&d_data[f*n_beams*n_time_pol_out + o*n_timesteps*n_pol], n_beams,
	// 						&d_data2[f*n_beams*n_time_pol_out + o*n_timesteps*n_pol], n_beams,
	// 						&detect[f*n_outputs_per_input*o]));

	std::cout << " data = " << detect[f*n_outputs_per_input*o].x <<std::endl;

	// gpuBLASchk(hipblasDotcEx(handle,
	// 						n_time_pol_out,
	// 						&d_data[f*n_beams*n_time_pol_out + o*n_timesteps*n_pol],
	// 						HIP_C_32F,
	// 						n_beams,
	// 						&d_data2[f*n_beams*n_time_pol_out + o*n_timesteps*n_pol],
	// 						HIP_C_32F,
	// 						n_beams,
	// 						&detect[f*n_outputs_per_input+o],
	// 						HIP_C_32F,
	// 						HIP_C_32F));

	// for (int f = 0; f < n_freq; f++){
	// 	for (int o = 0; o < n_outputs_per_input; o++){
	// 		std::cout << "f = " << f << " o = " << o << std::endl;

	// 	}
	// }




	gpuErrchk(hipFree(d_data));
	gpuErrchk(hipFree(detect));
	gpuErrchk(hipFree(d_data2));

	gpuBLASchk(hipblasDestroy(handle));

	delete[] data;
	return 0;
}



