
#include <hip/hip_runtime.h>
#include <iostream>

int main(){

	std::cout << "hi" << std::endl;

	int devicesCount;
	hipGetDeviceCount(&devicesCount);
	for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
	{
	    hipDeviceProp_t deviceProperties;
	    hipGetDeviceProperties(&deviceProperties, deviceIndex);
	//     printf("Device name: %s", deviceProperties.name);
	    std::cout << deviceProperties.name << std::endl;
	}

	return 0;

}