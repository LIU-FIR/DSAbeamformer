#include "hip/hip_runtime.h"
#include <iostream>
// #include <>

#define N_BEAMS 256
#define N_AVERAGING 16
#define N_TIMESTEPS_PER_CALL 64
#define N_POL 2
#define N 1000


typedef char2 CxInt8_t;

//Typedefs to maximize bandwidth usage
typedef char[4] c4; //32-bit so global memory bandwidth usage is optimal
typedef char[8] c8; //64-bit so global memory bandwidth usage is optimal


union doubleType{
	char a[2];
	int16_t b;
};

union quad_t{
	char a[4];
	int16_t b[2];
	CxInt8_t c[2];
};

__global__
void reduce_input(c4 *input, c8 *output, int input_size){
	'''
	This code takes in an array of 4-bit integers and returns an array of 8-bit integers.
	To maximize global memory bandwidth and symplicity, two special char types are 
	defined: c4 and c8. The size of these types are 32-bit and 64-bit respectively. These
	enable coalesced memory accesses, but then require the gpu to handle the 4 to 8-bit
	conversion simultaneously for 8 numbers (4 real/imaginary pairs). 

	'''
	__shared__ c4 shmem_in[32];
	__shared__ c8 shmem_out[32];

	int local_idx = threadIdx.x;
	int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

	while (global_idx < input_size){
		shmem_in[local_idx] = input[global_idx]; // read eight pieces of 4-bit memory into shared memory

		#pragma unroll
		for (int i = 0; i < 4; i++){
			char temp = shmem_in[local_idx][i];

			// break the char into two 4-bit chunks, then convert to 8-bit
			char high = (temp >> 4); // roll the 		most  significant 4 bits over the least significant 4 bits
			char low = (temp << 4);  // roll the 		least significant 4 bits over the most  significant 4 bits
			low = (low >> 4); 	 	 // roll the *new* 	most  significant 4 bits over the least significant 4 bits

			// store the two 8-bit numbers to the output shared memory array
			shmem_out[local_idx][2*i + 0] = high; 
			shmem_out[local_idx][2*i + 1] = low;
		}

		output[global_idx] = shmem_out[local_idx];	// write eight pieces of 8-bit memory out to global memory

		global_idx += gridDim.x *blockDim.x;
	}
}


__global__
void detect_sum(CuComplex *input, CuComplex *output){
	__shared__ CuComplex shmem[N_BEAMS];

	int tid = blockIdx.x * blockDim.x*N_POL*N_AVERAGING + threadIdx.x;

	hipComplex in;

	#pragma unroll
	for (int i = 0; i < N_POL*N_AVERAGING){
		in = input[tid];
		shmem[tid] += in*in;
		tid += blockDim.x;
	}



}



int main (){
	std::cout << "hello " << std::endl;



	char *in = new char[N];
	doubleType *out = new doubleType[N];

	for (int i = 0; i < N; i ++){
		in[i] = 0xE7;
	}

	char *d_in;
	doubleType *d_out;

	hipMalloc(&d_in, N*sizeof(char));
	hipMalloc(&d_out, N*sizeof(doubleType));

	hipMemcpy(d_in, in, N*sizeof(char), hipMemcpyHostToDevice);

	reduce_input<<<20,32>>>(d_in, d_out);

	hipMemcpy(d_out, out, N*sizeof(doubleType), hipMemcpyDeviceToHost);

	for (int i = 0;)





	return 0;
}



