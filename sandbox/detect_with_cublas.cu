#include <iostream>
#include <hipblas.h>
#include "nvToolsExt.h"
// #include <hip/hip_runtime.h>

#define PUSH_NVTX_RANGE(name,cid)  \
    int color_id = cid; \
    color_id = color_id%num_colors;\
    nvtxEventAttributes_t eventAttrib = {0}; \
    eventAttrib.version = NVTX_VERSION; \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
    eventAttrib.colorType = NVTX_COLOR_ARGB; \
    eventAttrib.color = colors[color_id]; \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
    eventAttrib.message.ascii = name; \
    nvtxRangePushEx(&eventAttrib); \



    
// hipEvent_t start;
// hipEvent_t stop;

// #define START_TIMER() {                         \
// 	gpuErrchk(hipEventCreate(&start));       \
// 	gpuErrchk(hipEventCreate(&stop));        \
// 	gpuErrchk(hipEventRecord(start));        \
// }

// #define STOP_RECORD_TIMER(name) {                           \
// 	gpuErrchk(hipEventRecord(stop));                     \
// 	gpuErrchk(hipEventSynchronize(stop));                \
// 	gpuErrchk(hipEventElapsedTime(&name, start, stop));  \
// 	gpuErrchk(hipEventDestroy(start));                   \
// 	gpuErrchk(hipEventDestroy(stop));                    \
// }

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

void gpuBLASchk(int errval){
	if (errval != HIPBLAS_STATUS_SUCCESS){
		std::cerr << "Failed BLAS call, error code " << errval << std::endl;
	}
}


int main(){
	
	std::cout << "hello " << std::endl;

	hipblasHandle_t handle;
	gpuBLASchk(hipblasCreate(&handle));
	gpuBLASchk(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE));


	int N =10;
	std::cout  << "N = " << N << std::endl;


	hipComplex *data = new hipComplex[N];

	for (int i = 0; i < N; i++){
		data[i].y = 0;
		data[i].x = i + .5;
	}

	for (int i = 0; i < 10; i ++){
		std::cout << "x[" << i << "] = " << data[i].x << " + " << data[i].y << "j" << std::endl;
	}
	std::cout << std::endl;

	hipComplex *d_data, *d_data2, *detect;

	gpuErrchk(hipMalloc(&d_data, N*sizeof(hipComplex)));
	gpuErrchk(hipMalloc(&d_data2, N*sizeof(hipComplex)));
	gpuErrchk(hipMalloc(&detect, 1*sizeof(hipComplex)));

	gpuErrchk(hipMemcpy(d_data, data, N*sizeof(hipComplex), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_data2, data, N*sizeof(hipComplex), hipMemcpyHostToDevice));


	hipComplex ans;

	gpuBLASchk(hipblasCdotc(handle, N,
							d_data, 1,
							d_data2, 1,
							detect));

	std::cout << "done" << std::endl;
	// gpuBLASchk(hipblasCdotc(handle, n_timesteps*n_pol,
	// 						&d_data[f*n_beams*n_time_pol_out + o*n_timesteps*n_pol], n_beams,
	// 						&d_data2[f*n_beams*n_time_pol_out + o*n_timesteps*n_pol], n_beams,
	// 						&detect[f*n_outputs_per_input*o]));
	gpuErrchk(hipMemcpy(&ans, detect, 1*sizeof(hipComplex), hipMemcpyDeviceToHost));
	std::cout << " data = " << ans.x <<std::endl;




	gpuErrchk(hipFree(d_data));
	gpuErrchk(hipFree(detect));
	gpuErrchk(hipFree(d_data2));

	gpuBLASchk(hipblasDestroy(handle));

	delete[] data;
	return 0;
}






