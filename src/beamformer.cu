#include "hip/hip_runtime.h"
#include "beamformer.cuh"



int main(){
	std::cout << "hello" << std::endl;

	std::ofstream f;
	f.open("bin/data.py");
	f << "A = [[";

	int N_DIRS = 1024;

	int A_rows	 = N_BEAMS;
	int A_cols 	 = N_ANTENNAS;
	int A_stride = A_rows*A_cols;
	int B_cols	 = N_TIMESTEPS_PER_CALL;
	int B_rows	 = A_cols;
	int B_stride = B_rows*B_cols;
	int C_rows	 = A_rows;
	int C_cols	 = B_cols;
	int C_stride = C_rows*C_cols;
	float bw_per_channel = (END_F - START_F)/TOT_CHANNELS; 

	/* GPU Variables */
	CxInt8_t *d_A; 				// Weight matrix (N_BEAMS X N_ANTENNAS, for N_FREQUENCIES)
	CxInt8_t *d_B; 				// Data Matrix (N_ANTENNAS X N_TIMESTEPS_PER_CALL, for N_FREQUENCIES)
	char *d_data;			// Raw input data (Before data massaging)
	hipComplex *d_C;				// Beamformed output (N_BEAMS X N_TIMESTEPS_PER_CALL, for N_FREQUENCIES)
	float *d_out;			// Data after being averaged over 16 time samples and 2 polarizations
	float *d_dedispersed;	// Data after being de-dispersed
	float *d_vec_ones;		// Vector of all ones for de-dispersion

	/* HOST Variables */
	CxInt8_t *A = new CxInt8_t[A_cols*A_rows*N_FREQUENCIES];
	CxInt8_t *B = new CxInt8_t[B_cols*B_rows*N_FREQUENCIES];
	char *data = new char[BYTES_PER_GEMM*N_BLOCKS_on_GPU]; //should be the size of one "dada block", data is 4-bit so real/imag is packed into one 8-bit char
	float *out_dedispersed = new float[N_BEAMS];
	float *vec_ones = new float[N_FREQUENCIES];

	// thrust::constant_iterator<float> vec_o(1)

	float* pos = new float[N_ANTENNAS];		// Locations of antennas
	float* dir = new float[N_BEAMS];		// Direction of beams
	int gpu = 0;							// Unique identifier for each GPU


	/* Populate location/direction Matricies */
	for (int i = 0; i < N_ANTENNAS; i++){
		pos[i] = i*500.0/(N_ANTENNAS-1) - 250.0;
	}

	/* Directions for Beamforming */
	for (int i = 0; i < N_BEAMS; i++){
		dir[i] = i*DEG2RAD(7.0)/(N_BEAMS-1) - DEG2RAD(3.5);
	}

	/* Create vector of ones for Dedispersion */
	for (int i = 0; i < N_FREQUENCIES; i++){
		vec_ones[i] = 1.0;
	}


	// Fourier Coefficient Matrix
	for (int i = 0; i < N_FREQUENCIES; i++){
		float freq = END_F - (ZERO_PT + gpu*TOT_CHANNELS/(N_GPUS-1) + i)*bw_per_channel;
		float wavelength = C_SPEED/(1E9*freq);
		for (int j = 0; j < N_ANTENNAS; j++){
			for (int k = 0; k < N_BEAMS; k++){
				A[i*A_stride + j*N_BEAMS + k].x = round(MAX_VAL*cos(-2*PI*pos[j]*sin(dir[k])/wavelength));
				A[i*A_stride + j*N_BEAMS + k].y = round(MAX_VAL*sin(-2*PI*pos[j]*sin(dir[k])/wavelength));
			}
		}
		// std::cout << "A[] = " << (int) A[i*N_ANTENNAS*N_BEAMS].x << "+"<< (int) A[i*N_ANTENNAS*N_BEAMS].y << "j" << std::endl;
	}

	// Signal Matrix
	// int test_frequency = 10;
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	/* Allocate and Move Memory to Device */
	hipMalloc(&d_A, 	A_rows*A_cols*N_FREQUENCIES*sizeof(CxInt8_t));
	hipMalloc(&d_B, 	B_rows*B_cols*N_FREQUENCIES*sizeof(CxInt8_t));
	hipMalloc(&d_C, 	C_rows*C_cols*N_FREQUENCIES*sizeof(hipComplex));
	hipMalloc(&d_data, BYTES_PER_GEMM*N_BLOCKS_on_GPU);
	hipMalloc(&d_out,  N_BEAMS*N_FREQUENCIES * sizeof(float));
	hipMalloc(&d_dedispersed, N_BEAMS*sizeof(float));
	hipMalloc(&d_vec_ones, N_BEAMS*sizeof(float));


	hipMemcpy(d_A, A, A_rows*A_cols*N_FREQUENCIES*sizeof(CxInt8_t), hipMemcpyHostToDevice);
	hipMemcpy(d_vec_ones, vec_ones, N_FREQUENCIES*sizeof(float), hipMemcpyHostToDevice);


	for (int iii = 0; iii < N_DIRS; iii++){
		float test_direction = DEG2RAD(-3.5) + iii*DEG2RAD(7.0)/(N_DIRS-1);

		// for (int i = 0; i < N_FREQUENCIES; i++){
		// 	float freq = END_F - (ZERO_PT + gpu*TOT_CHANNELS/(N_GPUS-1) + i)*bw_per_channel;
		// 	float wavelength = C_SPEED/(1E9*freq);

		// 	for (int j = 0; j < N_TIMESTEPS_PER_CALL; j++){
		// 		for (int k = 0; k < N_ANTENNAS; k++){
		// 			if (i == test_frequency){
		// 				B[i*N_TIMESTEPS_PER_CALL*N_ANTENNAS + j*N_ANTENNAS + k].x = round(MAX_VAL*cos(2*PI*pos[k]*sin(test_direction)/wavelength));
		// 				B[i*N_TIMESTEPS_PER_CALL*N_ANTENNAS + j*N_ANTENNAS + k].y = round(MAX_VAL*sin(2*PI*pos[k]*sin(test_direction)/wavelength));
		// 			}
		// 		}
		// 	}

		// }
	 	
		// int simulated_direction = 100;
		int current_block = 0;
		// int tot_avging = N_POL*N_AVERAGING;

		char high, low;

		for (int i = 0; i < N_FREQUENCIES; i++){
			float freq = END_F - (ZERO_PT + gpu*TOT_CHANNELS/(N_GPUS-1) + i)*bw_per_channel;
			// std::cout << "freq: " << freq << std::endl;
			float wavelength = C_SPEED/(1E9*freq);
			for (int j = 0; j < N_TIMESTEPS_PER_CALL; j++){
				for (int k = 0; k < N_ANTENNAS; k++){

					high = ((char) round(SIG_MAX_VAL*cos(2*PI*pos[k]*sin(test_direction)/wavelength))); //real
					low  = ((char) round(SIG_MAX_VAL*sin(2*PI*pos[k]*sin(test_direction)/wavelength))); //imag

					data[i*B_stride + j*N_ANTENNAS + k] = (high << 4) | (0x0F & low);
				}
			}
		}



		// hipMemcpy(d_B, B, B_rows*B_cols*N_FREQUENCIES*sizeof(CxInt8_t), hipMemcpyHostToDevice);
		hipMemcpy(&(d_data[BYTES_PER_GEMM*current_block]), data, BYTES_PER_GEMM, hipMemcpyHostToDevice);

		
		expand_input<<<1000, 32>>>(d_data, (char *) d_B, B_stride*N_FREQUENCIES);



		// Multiplicative Constants
		hipComplex inv_max_value, zero;//, one;
		inv_max_value.x = 1.0/MAX_VAL;
		inv_max_value.y = 0;
		zero.x = 0;
		zero.y = 0;
		// one.x = 1;
		// one.y = 0;

		hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
									A_rows, B_cols, A_cols,
									&inv_max_value,
									d_A, HIP_C_8I, A_rows, A_stride,
									d_B, HIP_C_8I, B_rows, B_stride,
									&zero,
									d_C, HIP_C_32F, C_rows, C_stride,
									N_FREQUENCIES, HIP_C_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);


		detect_sum<<<N_FREQUENCIES, N_BEAMS>>>(d_C, d_out);

		float f_one = 1.0;
		float f_zero = 0.0;


		hipblasSgemv(handle, HIPBLAS_OP_N,
					N_BEAMS, N_FREQUENCIES,
					&f_one,
					d_out, N_BEAMS,
					d_vec_ones, 1,
					&f_zero,
					d_dedispersed, 1);


		//gemv to dedisperse
		//copy to host
		//sfml image

		hipMemcpy(out_dedispersed, d_dedispersed, N_BEAMS*sizeof(float), hipMemcpyDeviceToHost);

		for (int i = 0; i < N_BEAMS; i++){
			f << out_dedispersed[i];
			if (i != N_BEAMS - 1){
				f << ",";
			}
		}

		if (iii != N_DIRS-1){
			f << "],\n[";
		} else {
			f<< "]]";
		}

	}

	#if 0
		float max = 0;
		float rms = 0.0;
		int max_i = 0;
		for (int i = 0; i < 256; i++){
			if (C[i].x>max){
				rms += C[i].x*C[i].x;
				max = C[i].x;
				max_i = i;
			}
			std::cout << "C[" << i <<"] = " << C[i].x << "+" << C[i].y << "j" << std::endl;
		}

		std::cout << "max(C) = " << max_i << ", " << max << std::endl;
		std::cout << "rms(c) = " << sqrt(rms/256.0) << std::endl;
	#endif


	f.close();


	hipFree(d_A);
	hipFree(d_C);
	hipFree(d_B);
	hipFree(d_data);
	hipFree(d_out);
	hipFree(d_dedispersed);
	hipFree(d_vec_ones);

	delete[] vec_ones;
	delete[] A;
	delete[] out_dedispersed;
	delete[] data;
	delete[] B;
	delete[] pos;
	delete[] dir;

	hipblasDestroy(handle);
	return 0;
}