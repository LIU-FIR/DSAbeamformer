
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hipblas.h>



//https://solarianprogrammer.com/2012/05/31/matrix-multiplication-cuda-cublas-curand-thrust/
//nvcc mm.cu -lcublas -lcurand -o mm1


void GPU_fill_rand(float *A, int nr_rows, int nr_cols){
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
	hiprandGenerateUniform(prng, A, nr_rows*nr_cols);
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}


void gpuBLASchk(int errval){
	if (errval != HIPBLAS_STATUS_SUCCESS){
		std::cerr << "Failed BLAS call, error code " << errval << std::endl;
	}
}

hipEvent_t start;
hipEvent_t stop;

#define START_TIMER() {                         \
	gpuErrchk(hipEventCreate(&start));       \
	gpuErrchk(hipEventCreate(&stop));        \
	gpuErrchk(hipEventRecord(start));        \
}

#define STOP_RECORD_TIMER(name) {                           \
	gpuErrchk(hipEventRecord(stop));                     \
	gpuErrchk(hipEventSynchronize(stop));                \
	gpuErrchk(hipEventElapsedTime(&name, start, stop));  \
	gpuErrchk(hipEventDestroy(start));                   \
	gpuErrchk(hipEventDestroy(stop));                    \
}


int main(int argc, char const *argv[]){

	// std::cout << "matrix mult" << std::endl;

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	float *d_A, *d_B, *d_C;

	long na, nb, nmax;
	if (argc == 2){
		nmax = (long) atoi(argv[1]);
		na = 512;
		nb = 512;
	} else if (argc == 4){
		na = (long) atoi(argv[1]);
		nb = (long) atoi(argv[2]);
		nmax = (long) atoi(argv[3]);
	}
	std::cout << na << "," << nb << "," << nmax << ",";

	hipMalloc(&d_A, na*nmax * sizeof(float));
	hipMalloc(&d_B, nmax*nb * sizeof(float));
	hipMalloc(&d_C, na*nb * sizeof(float));

	GPU_fill_rand(d_A, na, nmax);
	GPU_fill_rand(d_B, nmax, nb);
	GPU_fill_rand(d_C, na, nb);

	float alf = 1.0;
	float bet = 0.0;

	float ops;
	int k_avg = 5;

	for (int n = 1; n < nmax; n++){
		float timer = 0;
		float tot = 0;
		for (int k = 0; k < k_avg; k++){
			
			hipEventCreate			gpuBLASchk(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, na, nb, n, &alf, d_A, na, d_B, n, &bet, d_C, na));
			STOP_RECORD_TIMER(timer);
			tot += timer;
		}

		ops = (float) na*nb;
		ops *= 2*n-1;

		std::cout << ops/tot*1000.0*(k_avg);
		if( n != nmax -1 ){
			std::cout << ",";
		}
		//<< "FLOP = " << n*n*(2*n-1) << ", Time = " << timer << " ms, FLOPS = "
	}

	std::cout << std::endl;

	return 0;

}


